
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define SOFTENING 1e-9f
#define nBodies 120000

typedef struct { float4 *pos, *vel; } BodySystem;

double get_time() // function to get the time of day in seconds
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + tv.tv_usec * 1e-6;
}

void randomizeBodies(float *data, int n) {
  for (int i = 0; i < n; i++) {
    data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
  }
}

__global__
void bodyForce(float4 *p, float4 *v, float dt, int n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;

    for (int tile = 0; tile < gridDim.x; tile++) {
      extern __shared__ float3 spos[];
      float4 tpos = p[tile * blockDim.x + threadIdx.x];
      spos[threadIdx.x] = make_float3(tpos.x, tpos.y, tpos.z);
      __syncthreads();

      for (int j = 0; j < blockDim.x; j++) {
        float dx = spos[j].x - p[i].x;
        float dy = spos[j].y - p[i].y;
        float dz = spos[j].z - p[i].z;
        float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
        float invDist = rsqrtf(distSqr);
        float invDist3 = invDist * invDist * invDist;

        Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
      }
      __syncthreads();
    }

    v[i].x += dt*Fx; v[i].y += dt*Fy; v[i].z += dt*Fz;
  }
}

int main(const int argc, const char** argv) {
  
  
  if (argc < 2){
    printf("Usage: ./exec BLOCK_SIZE\n");
    return 0;
  }

  int BLOCK_SIZE = atoi(argv[1]);

  const float dt = 0.01f; // time step
  const int nIters = 10;  // simulation iterations
  
  int bytes = 2*nBodies*sizeof(float4);
  float *buf = (float*)malloc(bytes);
  BodySystem p = { (float4*)buf, ((float4*)buf) + nBodies };

  randomizeBodies(buf, 8*nBodies); // Init pos / vel data

  float *d_buf;
  hipMalloc(&d_buf, bytes);
  BodySystem d_p = { (float4*)d_buf, ((float4*)d_buf) + nBodies };

  int nBlocks = (nBodies + BLOCK_SIZE - 1) / BLOCK_SIZE;
  double totalTime = 0.0; 

  for (int iter = 1; iter <= nIters; iter++) {
    double start_gpu = get_time();

    hipMemcpy(d_buf, buf, bytes, hipMemcpyHostToDevice);
    bodyForce<<<nBlocks, BLOCK_SIZE, BLOCK_SIZE*sizeof(float3)>>>(d_p.pos, d_p.vel, dt, nBodies);
    hipMemcpy(buf, d_buf, bytes, hipMemcpyDeviceToHost);

    for (int i = 0 ; i < nBodies; i++) { // integrate position
      p.pos[i].x += p.vel[i].x*dt;
      p.pos[i].y += p.vel[i].y*dt;
      p.pos[i].z += p.vel[i].z*dt;
    }

    double end_gpu = get_time();

    const double tElapsed = end_gpu-start_gpu;
    if (iter > 1) { // First iter is warm up
      totalTime += tElapsed; 
    }

    printf("Iteration %d: %.3f seconds\n", iter, tElapsed);

  }
  double avgTime = totalTime / (double)(nIters-1); 

  printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, 1e-9 * nBodies * nBodies / avgTime);

  free(buf);
  hipFree(d_buf);
}
