
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define SOFTENING 1e-9f
#define nBodies 120000

typedef struct { float x, y, z, vx, vy, vz; } Body;

double get_time() // function to get the time of day in seconds
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + tv.tv_usec * 1e-6;
}

void randomizeBodies(float *data, int n) {
  for (int i = 0; i < n; i++) {
    data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
  }
}

__global__
void bodyForce(Body *p, float dt, int n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;

    for (int j = 0; j < n; j++) {
      float dx = p[j].x - p[i].x;
      float dy = p[j].y - p[i].y;
      float dz = p[j].z - p[i].z;
      float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
      float invDist = rsqrtf(distSqr);
      float invDist3 = invDist * invDist * invDist;

      Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
    }

    p[i].vx += dt*Fx; p[i].vy += dt*Fy; p[i].vz += dt*Fz;
  }
}

int main(const int argc, const char** argv) {
  
  if (argc < 2){
    printf("Usage: ./exec BLOCK_SIZE\n");
    return 0;
  }

  int BLOCK_SIZE = atoi(argv[1]);
  
  const float dt = 0.01f; // time step
  const int nIters = 10;  // simulation iterations

  int bytes = nBodies*sizeof(Body);
  float *buf = (float*)malloc(bytes);
  Body *p = (Body*)buf;

  randomizeBodies(buf, 6*nBodies); // Init pos / vel data

  float *d_buf;
  hipMalloc(&d_buf, bytes);
  Body *d_p = (Body*)d_buf;

  int nBlocks = (nBodies + BLOCK_SIZE - 1) / BLOCK_SIZE;
  double totalTime = 0.0; 

  for (int iter = 1; iter <= nIters; iter++) {
    double start_gpu = get_time();

    hipMemcpy(d_buf, buf, bytes, hipMemcpyHostToDevice);
    bodyForce<<<nBlocks, BLOCK_SIZE>>>(d_p, dt, nBodies); // compute interbody forces
    hipMemcpy(buf, d_buf, bytes, hipMemcpyDeviceToHost);

    for (int i = 0 ; i < nBodies; i++) { // integrate position
      p[i].x += p[i].vx*dt;
      p[i].y += p[i].vy*dt;
      p[i].z += p[i].vz*dt;
    }

    double end_gpu = get_time();

    const double tElapsed = end_gpu - start_gpu;
    if (iter > 1) { // First iter is warm up
      totalTime += tElapsed; 
    }

    printf("Iteration %d: %.3f seconds\n", iter, tElapsed);

  }
  double avgTime = totalTime / (double)(nIters-1); 
  printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, 1e-9 * nBodies * nBodies / avgTime);

  free(buf);
  hipFree(d_buf);
}
